#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include <random>
#include <vector>
#include <string>
#include <algorithm>
#include <chrono>
#include <cstdio>

using namespace std;

int main() {
    const char* str1 = "ATCG";
    const char* str2 = "ATCD";

    size_t length = std::strlen(str1);

    // Alocar memória no device
    char* dev_str1;
    char* dev_str2;
    hipMalloc((void**)&dev_str1, length * sizeof(char));
    hipMalloc((void**)&dev_str2, length * sizeof(char));

    // Copiar dados para o device
    hipMemcpy(dev_str1, str1, length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_str2, str2, length * sizeof(char), hipMemcpyHostToDevice);

    // Criar thrust::device_ptr a partir dos ponteiros no device
    thrust::device_ptr dev_ptr_str1(dev_str1);
    thrust::device_ptr dev_ptr_str2(dev_str2);

    bool equal = thrust::equal(dev_ptr_str1, dev_ptr_str1 + length, dev_ptr_str2);

    if (equal) {
        std::cout << "As strings são iguais." << std::endl;
    } else {
        std::cout << "As strings são diferentes." << std::endl;
    }

    // Liberar memória no device
    hipFree(dev_str1);
    hipFree(dev_str2);

    return 0;
}