#include "hip/hip_runtime.h"
#include <iostream>
 #include <iomanip>
 #include <cstdlib>
 #include <chrono>
 #include <cstdlib>
 #include <algorithm>
 #include <thrust/host_vector.h>
 #include <thrust/device_vector.h>
 #include <thrust/generate.h>
 #include <thrust/functional.h>
 #include <thrust/copy.h>
 #include <thrust/transform.h>
 #include <cmath>
 #include <random>

 using namespace std::chrono;

 void reportTime(const char* msg, steady_clock::duration span) {
     auto ms = duration_cast<milliseconds>(span);
     std::cout << msg << " - levou - " <<
      ms.count() << " milisegundos" << std::endl;
 }

 // CRIE UMA FUNCTOR PARA CALCULAR A SQUARE

struct square
{    
    square() {};
    __host__ __device__
    double operator()(const double& x) {
           return x * x;
    }
};

 // IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
 float magnitude(thrust::device_vector<double> v) {
     float result;

    result = std::sqrt(thrust::transform_reduce( v.begin(), v.end(), square(), 0, thrust::plus<double>()));

     return result;
 }

 int main(int argc, char** argv) {
     if (argc != 2) {
         std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
         std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
         return 1;
     }
     int n = std::atoi(argv[1]); //numero de elementos
     steady_clock::time_point ts, te;


     // Faça um  vector em thrust 
    thrust::device_vector<double> v_d(n);

     // inicilize o  vector
     ts = steady_clock::now();

     std::generate(v_d.begin(), v_d.end(), std::rand);


     te = steady_clock::now();
     reportTime("Inicializacao", te - ts);

     // Calcule a magnitude do vetor
     ts = steady_clock::now();
     float len = magnitude(v_d);
     te = steady_clock::now();
     reportTime("Tempo para calculo", te - ts);


     std::cout << std::fixed << std::setprecision(4);
     std::cout << "Magnitude : " << len << std::endl;
 }